
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    // kernel function name <<< number of block, number of thread >>> (arguments)
    cuda_hello<<<1,1>>>();
    
    // https://qiita.com/JmpM/items/ada670ec80be9566269e
    // CPU waits for GPU operation
    hipDeviceSynchronize();
    return 0;
}